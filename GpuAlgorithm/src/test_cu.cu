
//#include "../include/clion_cuda.h"
#include <iostream>

#include <stdio.h>
#include <hip/hip_runtime.h>


__global__
void add_f(float *array) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	array[k] = 10.0;
}


int main() {

	int blocks = 1024;
	int thread_pre_blocks = 30;
	float *array_host = new float[blocks * thread_pre_blocks];
	float *array_device;
	std::cout << "before" << std::endl;
	hipMalloc((void**)&array_device, blocks * thread_pre_blocks);

	add_f<<<blocks,thread_pre_blocks>>>(array_device);
	
	hipMemcpy(array_device, array_host, blocks * thread_pre_blocks, hipMemcpyDeviceToHost);


	for (int i(0); i < blocks * thread_pre_blocks; ++i) {
		std::cout << array_host[i] << std::endl;
	}
	hipFree(array_device);

}
