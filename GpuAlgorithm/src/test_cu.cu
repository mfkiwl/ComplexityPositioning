#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "../include/clion_cuda.h"

__device__ float add(float *array){
	const int k = blockIdx.x * blockDim.x + threadIdx.x;

	array[k] = k;
}


int main(){

	int blocks = 1024;
	int thread_pre_blocks = 30;
	float *array_host = new float[blocks*thread_pre_blocks];
	float *array_device;
	hipMalloc((void**)&array_device,blocks*thread_pre_blocks);
	add<<<blocks,thread_pre_blocks>>>(array_device);
	hipMemcpy(array_host,array_device,hipMemcpyDeviceToHost);
	for(int i(0);i<blocks*thread_pre_blocks;++i){
		std::cout << array_host[i] << std::endl;
	}
	hipFree(array_device);

}
